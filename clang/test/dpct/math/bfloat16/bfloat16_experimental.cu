// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.1, v11.2, v11.3, v11.4, v11.5
// RUN: dpct --format-range=none --use-experimental-features=bfloat16_math_functions -out-root %T/math/bfloat16/bfloat16_experimental %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/math/bfloat16/bfloat16_experimental/bfloat16_experimental.dp.cpp


#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"

__global__ void kernelFuncBfloat16Arithmetic() {
  // CHECK: sycl::ext::oneapi::bfloat16 bf16, bf16_1, bf16_2, bf16_3;
  __hip_bfloat16 bf16, bf16_1, bf16_2, bf16_3;
  // CHECK: bf16 = sycl::ext::oneapi::experimental::fabs(bf16_1);
  bf16 = __habs(bf16_1);
  // CHECK: bf16 = bf16_1 + bf16_2;
  bf16 = __hadd(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 + bf16_2;
  bf16 = __hadd_rn(bf16_1, bf16_2);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(bf16_1 + bf16_2, 0.f, 1.0f);
  bf16 = __hadd_sat(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 / bf16_2;
  bf16 = __hdiv(bf16_1, bf16_2);
  // CHECK: bf16 = sycl::ext::oneapi::experimental::fma(bf16_1, bf16_2, bf16_3);
  bf16 = __hfma(bf16_1, bf16_2, bf16_3);
  // CHECK: bf16 = dpct::relu(sycl::ext::oneapi::experimental::fma(bf16_1, bf16_2, bf16_3));
  bf16 = __hfma_relu(bf16_1, bf16_2, bf16_3);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(sycl::ext::oneapi::experimental::fma(bf16_1, bf16_2, bf16_3), 0.f, 1.0f);
  bf16 = __hfma_sat(bf16_1, bf16_2, bf16_3);
  // CHECK: bf16 = bf16_1 * bf16_2;
  bf16 = __hmul(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 * bf16_2;
  bf16 = __hmul_rn(bf16_1, bf16_2);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(bf16_1 * bf16_2, 0.f, 1.0f);
  bf16 = __hmul_sat(bf16_1, bf16_2);
  // CHECK: bf16 = -bf16_1;
  bf16 = __hneg(bf16_1);
  // CHECK: bf16 = bf16_1 - bf16_2;
  bf16 = __hsub(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 - bf16_2;
  bf16 = __hsub_rn(bf16_1, bf16_2);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(bf16_1 - bf16_2, 0.f, 1.0f);
  bf16 = __hsub_sat(bf16_1, bf16_2);
}

__global__ void kernelFuncBfloat162Arithmetic() {
  // CHECK: sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1, bf162_2, bf162_3;
  __hip_bfloat162 bf162, bf162_1, bf162_2, bf162_3;
  // CHECK: bf162 = bf162_1 / bf162_2;
  bf162 = __h2div(bf162_1, bf162_2);
  // CHECK: bf162 = sycl::ext::oneapi::experimental::fabs(bf162_1);
  bf162 = __habs2(bf162_1);
  // CHECK: bf162 = bf162_1 + bf162_2;
  bf162 = __hadd2(bf162_1, bf162_2);
  // CHECK: bf162 = bf162_1 + bf162_2;
  bf162 = __hadd2_rn(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::clamp(bf162_1 + bf162_2, {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hadd2_sat(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::complex_mul_add(bf162_1, bf162_2, bf162_3);
  bf162 = __hcmadd(bf162_1, bf162_2, bf162_3);
  // CHECK: bf162 = sycl::ext::oneapi::experimental::fma(bf162_1, bf162_2, bf162_3);
  bf162 = __hfma2(bf162_1, bf162_2, bf162_3);
  // CHECK: bf162 = dpct::relu(sycl::ext::oneapi::experimental::fma(bf162_1, bf162_2, bf162_3));
  bf162 = __hfma2_relu(bf162_1, bf162_2, bf162_3);
  // CHECK: bf162 = dpct::clamp(sycl::ext::oneapi::experimental::fma(bf162_1, bf162_2, bf162_3), {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hfma2_sat(bf162_1, bf162_2, bf162_3);
  // CHECK: bf162 = bf162_1 * bf162_2;
  bf162 = __hmul2(bf162_1, bf162_2);
  // CHECK: bf162 = bf162_1 * bf162_2;
  bf162 = __hmul2_rn(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::clamp(bf162_1 * bf162_2, {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hmul2_sat(bf162_1, bf162_2);
  // CHECK: bf162 = -bf162_1;
  bf162 = __hneg2(bf162_1);
  // CHECK: bf162 = bf162_1 - bf162_2;
  bf162 = __hsub2(bf162_1, bf162_2);
  // CHECK: bf162 = bf162_1 - bf162_2;
  bf162 = __hsub2_rn(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::clamp(bf162_1 - bf162_2, {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hsub2_sat(bf162_1, bf162_2);
}

__global__ void kernelFuncBfloat16Math() {
  // CHECK: sycl::ext::oneapi::bfloat16 bf16, bf16_1;
  __hip_bfloat16 bf16, bf16_1;
  // CHECK: bf16_1 = sycl::ext::oneapi::experimental::ceil(bf16);
  bf16_1 = hceil(bf16);
  // CHECK: bf16_1 = sycl::ext::oneapi::experimental::cos(bf16);
  bf16_1 = hcos(bf16);
  // CHECK: bf16_1 = sycl::ext::oneapi::experimental::exp(bf16);
  bf16_1 = hexp(bf16);
  // CHECK: bf16_1 = sycl::ext::oneapi::experimental::exp10(bf16);
  bf16_1 = hexp10(bf16);
  // CHECK: bf16_1 = sycl::ext::oneapi::experimental::exp2(bf16);
  bf16_1 = hexp2(bf16);
  // CHECK: bf16_1 = sycl::ext::oneapi::experimental::floor(bf16);
  bf16_1 = hfloor(bf16);
  // CHECK: bf16_1 = sycl::ext::oneapi::experimental::log(bf16);
  bf16_1 = hlog(bf16);
  // CHECK: bf16_1 = sycl::ext::oneapi::experimental::log10(bf16);
  bf16_1 = hlog10(bf16);
  // CHECK: bf16_1 = sycl::ext::oneapi::experimental::log2(bf16);
  bf16_1 = hlog2(bf16);
  // CHECK: bf16_1 = sycl::half_precision::recip(float(bf16));
  bf16_1 = hrcp(bf16);
  // CHECK: bf16_1 = sycl::ext::oneapi::experimental::rint(bf16);
  bf16_1 = hrint(bf16);
  // CHECK: bf16_1 = sycl::ext::oneapi::experimental::rsqrt(bf16);
  bf16_1 = hrsqrt(bf16);
  // CHECK: bf16_1 = sycl::ext::oneapi::experimental::sin(bf16);
  bf16_1 = hsin(bf16);
  // CHECK: bf16_1 = sycl::ext::oneapi::experimental::sqrt(bf16);
  bf16_1 = hsqrt(bf16);
  // CHECK: bf16_1 = sycl::ext::oneapi::experimental::trunc(bf16);
  bf16_1 = htrunc(bf16);
}

__global__ void kernelFuncBfloat162Math() {
  // CHECK: sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1;
  __hip_bfloat162 bf162, bf162_1;
  // CHECK: bf162_1 = sycl::ext::oneapi::experimental::ceil(bf162);
  bf162_1 = h2ceil(bf162);
  // CHECK: bf162_1 = sycl::ext::oneapi::experimental::cos(bf162);
  bf162_1 = h2cos(bf162);
  // CHECK: bf162_1 = sycl::ext::oneapi::experimental::exp(bf162);
  bf162_1 = h2exp(bf162);
  // CHECK: bf162_1 = sycl::ext::oneapi::experimental::exp10(bf162);
  bf162_1 = h2exp10(bf162);
  // CHECK: bf162_1 = sycl::ext::oneapi::experimental::exp2(bf162);
  bf162_1 = h2exp2(bf162);
  // CHECK: bf162_1 = sycl::ext::oneapi::experimental::floor(bf162);
  bf162_1 = h2floor(bf162);
  // CHECK: bf162_1 = sycl::ext::oneapi::experimental::log(bf162);
  bf162_1 = h2log(bf162);
  // CHECK: bf162_1 = sycl::ext::oneapi::experimental::log10(bf162);
  bf162_1 = h2log10(bf162);
  // CHECK: bf162_1 = sycl::ext::oneapi::experimental::log2(bf162);
  bf162_1 = h2log2(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::half_precision::recip(float(bf162[0])), sycl::half_precision::recip(float(bf162[1])));
  bf162_1 = h2rcp(bf162);
  // CHECK: bf162_1 = sycl::ext::oneapi::experimental::rint(bf162);
  bf162_1 = h2rint(bf162);
  // CHECK: bf162_1 = sycl::ext::oneapi::experimental::rsqrt(bf162);
  bf162_1 = h2rsqrt(bf162);
  // CHECK: bf162_1 = sycl::ext::oneapi::experimental::sin(bf162);
  bf162_1 = h2sin(bf162);
  // CHECK: bf162_1 = sycl::ext::oneapi::experimental::sqrt(bf162);
  bf162_1 = h2sqrt(bf162);
  // CHECK: bf162_1 = sycl::ext::oneapi::experimental::trunc(bf162);
  bf162_1 = h2trunc(bf162);
}

int main() { return 0; }
