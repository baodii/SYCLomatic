// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.1, v11.2, v11.3, v11.4, v11.5
// RUN: dpct --format-range=none -out-root %T/math/bfloat16/bfloat16 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/math/bfloat16/bfloat16/bfloat16.dp.cpp


#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"

// CHECK: class C : public sycl::marray<sycl::ext::oneapi::bfloat16, 2> {
class C : public __hip_bfloat162 {
  void f() {
    // CHECK: (*this)[0];
    // CHECK-NEXT: (*this)[1];
    x;
    y;
  }
};

// CHECK: void foo(sycl::ext::oneapi::bfloat16 *a, sycl::marray<sycl::ext::oneapi::bfloat16, 2> *b) {
void foo(__hip_bfloat16 *a, __hip_bfloat162 *b) {
  int i = 0;
  float f = 3.0f;
  // CHECK: a[i] = (sycl::ext::oneapi::bfloat16)f;
  a[i] = (__hip_bfloat16)f;

  // CHECK: (*b)[0];
  // CHECK-NEXT: (*b)[1];
  b->x;
  b->y;
}

__global__ void kernelFuncBfloat16Arithmetic() {
  // CHECK: sycl::ext::oneapi::bfloat16 bf16, bf16_1, bf16_2, bf16_3;
  __hip_bfloat16 bf16, bf16_1, bf16_2, bf16_3;
  // CHECK: bf16 = sycl::fabs(float(bf16_1));
  bf16 = __habs(bf16_1);
  // CHECK: bf16 = bf16_1 + bf16_2;
  bf16 = __hadd(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 + bf16_2;
  bf16 = __hadd_rn(bf16_1, bf16_2);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(bf16_1 + bf16_2, 0.f, 1.0f);
  bf16 = __hadd_sat(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 / bf16_2;
  bf16 = __hdiv(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 * bf16_2 + bf16_3;
  bf16 = __hfma(bf16_1, bf16_2, bf16_3);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hfma_relu is not supported.
  // CHECK-NEXT: */
  bf16 = __hfma_relu(bf16_1, bf16_2, bf16_3);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(bf16_1 * bf16_2 + bf16_3, 0.f, 1.0f);
  bf16 = __hfma_sat(bf16_1, bf16_2, bf16_3);
  // CHECK: bf16 = bf16_1 * bf16_2;
  bf16 = __hmul(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 * bf16_2;
  bf16 = __hmul_rn(bf16_1, bf16_2);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(bf16_1 * bf16_2, 0.f, 1.0f);
  bf16 = __hmul_sat(bf16_1, bf16_2);
  // CHECK: bf16 = -bf16_1;
  bf16 = __hneg(bf16_1);
  // CHECK: bf16 = bf16_1 - bf16_2;
  bf16 = __hsub(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 - bf16_2;
  bf16 = __hsub_rn(bf16_1, bf16_2);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(bf16_1 - bf16_2, 0.f, 1.0f);
  bf16 = __hsub_sat(bf16_1, bf16_2);
}

__global__ void kernelFuncBfloat162Arithmetic() {
  // CHECK: sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1, bf162_2, bf162_3;
  __hip_bfloat162 bf162, bf162_1, bf162_2, bf162_3;
  // CHECK: bf162 = bf162_1 / bf162_2;
  bf162 = __h2div(bf162_1, bf162_2);
  // CHECK: bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::fabs(float(bf162_1[0])), sycl::fabs(float(bf162_1[1])));
  bf162 = __habs2(bf162_1);
  // CHECK: bf162 = bf162_1 + bf162_2;
  bf162 = __hadd2(bf162_1, bf162_2);
  // CHECK: bf162 = bf162_1 + bf162_2;
  bf162 = __hadd2_rn(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::clamp(bf162_1 + bf162_2, {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hadd2_sat(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::complex_mul_add(bf162_1, bf162_2, bf162_3);
  bf162 = __hcmadd(bf162_1, bf162_2, bf162_3);
  // CHECK: bf162 = bf162_1 * bf162_2 + bf162_3;
  bf162 = __hfma2(bf162_1, bf162_2, bf162_3);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hfma2_relu is not supported.
  // CHECK-NEXT: */
  bf162 = __hfma2_relu(bf162_1, bf162_2, bf162_3);
  // CHECK: bf162 = dpct::clamp(bf162_1 * bf162_2 + bf162_3, {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hfma2_sat(bf162_1, bf162_2, bf162_3);
  // CHECK: bf162 = bf162_1 * bf162_2;
  bf162 = __hmul2(bf162_1, bf162_2);
  // CHECK: bf162 = bf162_1 * bf162_2;
  bf162 = __hmul2_rn(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::clamp(bf162_1 * bf162_2, {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hmul2_sat(bf162_1, bf162_2);
  // CHECK: bf162 = -bf162_1;
  bf162 = __hneg2(bf162_1);
  // CHECK: bf162 = bf162_1 - bf162_2;
  bf162 = __hsub2(bf162_1, bf162_2);
  // CHECK: bf162 = bf162_1 - bf162_2;
  bf162 = __hsub2_rn(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::clamp(bf162_1 - bf162_2, {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hsub2_sat(bf162_1, bf162_2);
}

// CHECK: void test_conversions_device(sycl::ext::oneapi::bfloat16 *deviceArrayBFloat16) {
// CHECK-NEXT:   float f, f_1, f_2;
// CHECK-NEXT:   sycl::float2 f2, f2_1, f2_2;
// CHECK-NEXT:   sycl::ext::oneapi::bfloat16 bf16, bf16_1, bf16_2;
// CHECK-NEXT:   sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1, bf162_2;
// CHECK-NEXT:   f2 = sycl::float2(bf162[0], bf162[1]);
// CHECK-NEXT:   f = static_cast<float>(bf16);
// CHECK-NEXT:   bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(f2[0], f2[1]);
// CHECK-NEXT:   bf16 = sycl::ext::oneapi::bfloat16(f);
__global__ void test_conversions_device(__hip_bfloat16 *deviceArrayBFloat16) {
  float f, f_1, f_2;
  float2 f2, f2_1, f2_2;
  __hip_bfloat16 bf16, bf16_1, bf16_2;
  __hip_bfloat162 bf162, bf162_1, bf162_2;
  f2 = __bfloat1622float2(bf162);
  f = __bfloat162float(bf16);
  bf162 = __float22bfloat162_rn(f2);
  bf16 = __float2bfloat16(f);

  // CHECK:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  bf16_2 = __ldca(deviceArrayBFloat16);
  bf16_2 = __ldca(&bf16);
  bf162_2 = __ldca(&bf162);
  bf16_2 = __ldcg(deviceArrayBFloat16);
  bf16_2 = __ldcg(&bf16);
  bf162_2 = __ldcg(&bf162);
  bf16_2 = __ldcs(deviceArrayBFloat16);
  bf16_2 = __ldcs(&bf16);
  bf162_2 = __ldcs(&bf162);
  bf16_2 = __ldcv(deviceArrayBFloat16);
  bf16_2 = __ldcv(&bf16);
  bf162_2 = __ldcv(&bf162);
  bf16_2 = __ldg(deviceArrayBFloat16);
  bf16_2 = __ldg(&bf16);
  bf162_2 = __ldg(&bf162);
  bf16_2 = __ldlu(deviceArrayBFloat16);
  bf16_2 = __ldlu(&bf16);
  bf162_2 = __ldlu(&bf162);

  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *(deviceArrayBFloat16 + 1) = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf16_2 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf162_2 = bf162;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *deviceArrayBFloat16 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf16_2 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf162_2 = bf162;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *(deviceArrayBFloat16 + 1) = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf16_2 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf162_2 = bf162;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *deviceArrayBFloat16 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf16_2 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf162_2 = bf162;
  __stcg(deviceArrayBFloat16 + 1, bf16);
  __stcg(&bf16_2, bf16);
  __stcg(&bf162_2, bf162);
  __stcs(deviceArrayBFloat16, bf16);
  __stcs(&bf16_2, bf16);
  __stcs(&bf162_2, bf162);
  __stwb(deviceArrayBFloat16 + 1, bf16);
  __stwb(&bf16_2, bf16);
  __stwb(&bf162_2, bf162);
  __stwt(deviceArrayBFloat16, bf16);
  __stwt(&bf16_2, bf16);
  __stwt(&bf162_2, bf162);
}

// CHECK: void test_conversions() {
// CHECK-NEXT:   float f, f_1, f_2;
// CHECK-NEXT:   sycl::float2 f2, f2_1, f2_2;
// CHECK-NEXT:   sycl::ext::oneapi::bfloat16 bf16, bf16_1, bf16_2;
// CHECK-NEXT:   sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1, bf162_2;
// CHECK-NEXT:   f2 = sycl::float2(bf162[0], bf162[1]);
// CHECK-NEXT:   f = static_cast<float>(bf16);
// CHECK-NEXT:   bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(f2[0], f2[1]);
// CHECK-NEXT:   bf16 = sycl::ext::oneapi::bfloat16(f);
void test_conversions() {
  float f, f_1, f_2;
  float2 f2, f2_1, f2_2;
  __hip_bfloat16 bf16, bf16_1, bf16_2;
  __hip_bfloat162 bf162, bf162_1, bf162_2;
  f2 = __bfloat1622float2(bf162);
  f = __bfloat162float(bf16);
  bf162 = __float22bfloat162_rn(f2);
  bf16 = __float2bfloat16(f);
}

__global__ void kernelFuncBfloat16Math() {
  // CHECK: sycl::ext::oneapi::bfloat16 bf16, bf16_1;
  __hip_bfloat16 bf16, bf16_1;
  // CHECK: bf16_1 = sycl::ceil(float(bf16));
  bf16_1 = hceil(bf16);
  // CHECK: bf16_1 = sycl::cos(float(bf16));
  bf16_1 = hcos(bf16);
  // CHECK: bf16_1 = sycl::exp(float(bf16));
  bf16_1 = hexp(bf16);
  // CHECK: bf16_1 = sycl::exp10(float(bf16));
  bf16_1 = hexp10(bf16);
  // CHECK: bf16_1 = sycl::exp2(float(bf16));
  bf16_1 = hexp2(bf16);
  // CHECK: bf16_1 = sycl::floor(float(bf16));
  bf16_1 = hfloor(bf16);
  // CHECK: bf16_1 = sycl::log(float(bf16));
  bf16_1 = hlog(bf16);
  // CHECK: bf16_1 = sycl::log10(float(bf16));
  bf16_1 = hlog10(bf16);
  // CHECK: bf16_1 = sycl::log2(float(bf16));
  bf16_1 = hlog2(bf16);
  // CHECK: bf16_1 = sycl::half_precision::recip(float(bf16));
  bf16_1 = hrcp(bf16);
  // CHECK: bf16_1 = sycl::rint(float(bf16));
  bf16_1 = hrint(bf16);
  // CHECK: bf16_1 = sycl::rsqrt(float(bf16));
  bf16_1 = hrsqrt(bf16);
  // CHECK: bf16_1 = sycl::sin(float(bf16));
  bf16_1 = hsin(bf16);
  // CHECK: bf16_1 = sycl::sqrt(float(bf16));
  bf16_1 = hsqrt(bf16);
  // CHECK: bf16_1 = sycl::trunc(float(bf16));
  bf16_1 = htrunc(bf16);
}

__global__ void kernelFuncBfloat162Math() {
  // CHECK: sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1;
  __hip_bfloat162 bf162, bf162_1;
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::ceil(float(bf162[0])), sycl::ceil(float(bf162[1])));
  bf162_1 = h2ceil(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::cos(float(bf162[0])), sycl::cos(float(bf162[1])));
  bf162_1 = h2cos(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::exp(float(bf162[0])), sycl::exp(float(bf162[1])));
  bf162_1 = h2exp(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::exp10(float(bf162[0])), sycl::exp10(float(bf162[1])));
  bf162_1 = h2exp10(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::exp2(float(bf162[0])), sycl::exp2(float(bf162[1])));
  bf162_1 = h2exp2(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::floor(float(bf162[0])), sycl::floor(float(bf162[1])));
  bf162_1 = h2floor(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::log(float(bf162[0])), sycl::log(float(bf162[1])));
  bf162_1 = h2log(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::log10(float(bf162[0])), sycl::log10(float(bf162[1])));
  bf162_1 = h2log10(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::log2(float(bf162[0])), sycl::log2(float(bf162[1])));
  bf162_1 = h2log2(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::half_precision::recip(float(bf162[0])), sycl::half_precision::recip(float(bf162[1])));
  bf162_1 = h2rcp(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::rint(float(bf162[0])), sycl::rint(float(bf162[1])));
  bf162_1 = h2rint(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::rsqrt(float(bf162[0])), sycl::rsqrt(float(bf162[1])));
  bf162_1 = h2rsqrt(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::sin(float(bf162[0])), sycl::sin(float(bf162[1])));
  bf162_1 = h2sin(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::sqrt(float(bf162[0])), sycl::sqrt(float(bf162[1])));
  bf162_1 = h2sqrt(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::trunc(float(bf162[0])), sycl::trunc(float(bf162[1])));
  bf162_1 = h2trunc(bf162);
}

int main() { return 0; }
