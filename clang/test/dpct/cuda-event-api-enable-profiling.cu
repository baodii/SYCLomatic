// RUN: dpct --enable-profiling --format-range=none --usm-level=none -out-root %T/cuda-event-api-enable-profiling %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cuda-event-api-enable-profiling/cuda-event-api-enable-profiling.dp.cpp --match-full-lines %s

// CHECK:#define DPCT_PROFILING_ENABLED
// CHECK-NEXT: #define DPCT_USM_LEVEL_NONE
// CHECK-NEXT: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <stdio.h>

#include <hip/hip_runtime.h>
#include <stdio.h>

template <typename T>
void my_error_checker(T ReturnValue, char const *const FuncName) {
}

#define MY_ERROR_CHECKER(CALL) my_error_checker((CALL), #CALL)

//CHECK: #define CudaEvent(X) X = new sycl::event()
#define CudaEvent(X) hipEventCreate(&X)

#define MY_CHECKER(CALL) do {                           \
  hipError_t Error = CALL;                             \
  if (Error != hipSuccess) {                           \
  }                                                     \
} while(0)

__global__ void kernelFunc()
{
}

int main(int argc, char* argv[]) {
// CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.out_of_order_queue();
// CHECK: dpct::event_ptr start, stop;
// CHECK-EMPTY:
// CHECK: start = new sycl::event();
// CHECK: stop = new sycl::event();
// CHECK-EMPTY:
// CHECK-NEXT: float elapsed_time;
// CHECK-EMPTY:
// CHECK-NEXT: dev_ct1.queues_wait_and_throw();
// CHECK-EMPTY:
// CHECK-NEXT: int blocks = 32, threads = 32;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  float elapsed_time;

  hipDeviceSynchronize();

  int blocks = 32, threads = 32;

// CHECK: printf("<<<\n");
// CHECK-NEXT: CudaEvent(start);
// CHECK-NEXT: printf(">>>\n");
  printf("<<<\n");
  CudaEvent(start);
  printf(">>>\n");


// CHECK: MY_ERROR_CHECKER(DPCT_CHECK_ERROR(start = new sycl::event()));
// CHECK: dpct::err0 et = DPCT_CHECK_ERROR(stop = new sycl::event());
  MY_ERROR_CHECKER(hipEventCreate(&start));
  hipError_t et = hipEventCreate(&stop);


  // kernel call without sync
// CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
// CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
// CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:           kernelFunc();
// CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();

// CHECK:   *start = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(start, 0);

  // kernel call without sync
// CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
// CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
// CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:           kernelFunc();
// CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();


// CHECK: *start = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(start, 0);

// CHECK: MY_ERROR_CHECKER(DPCT_CHECK_ERROR(*start = q_ct1.ext_oneapi_submit_barrier()));
  MY_ERROR_CHECKER(hipEventRecord(start, 0));

// CHECK: if (0)
// CHECK-NEXT:   /*
// CHECK-NEXT:   DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
// CHECK-NEXT:   */
// CHECK-NEXT:   MY_ERROR_CHECKER(DPCT_CHECK_ERROR(*start = q_ct1.ext_oneapi_submit_barrier()));
  if (0)
    MY_ERROR_CHECKER(hipEventRecord(start, 0));

  // kernel call with sync
// CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
// CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
// CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:           kernelFunc();
// CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();
// CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
// CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
// CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:           kernelFunc();
// CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();

// CHECK:   *stop = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(stop, 0);

// CHECK: MY_ERROR_CHECKER(DPCT_CHECK_ERROR(*stop = q_ct1.ext_oneapi_submit_barrier()));
  MY_ERROR_CHECKER(hipEventRecord(stop, 0));


// CHECK:   MY_ERROR_CHECKER(DPCT_CHECK_ERROR(*stop = q_ct1.ext_oneapi_submit_barrier()));
  if (1)
    MY_ERROR_CHECKER(hipEventRecord(stop, 0));

  // kernel call without sync
// CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
// CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
// CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:           kernelFunc();
// CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();

// CHECK: *stop = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(stop, 0);

// CHECK: /*
// CHECK-NEXT: DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
// CHECK-NEXT: */
// CHECK-NEXT: MY_ERROR_CHECKER(DPCT_CHECK_ERROR(*stop = q_ct1.ext_oneapi_submit_barrier()));
  MY_ERROR_CHECKER(hipEventRecord(stop, 0));

// CHECK: if (0)
// CHECK-NEXT:   /*
// CHECK-NEXT:   DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
// CHECK-NEXT:   */
// CHECK-NEXT:   MY_ERROR_CHECKER(DPCT_CHECK_ERROR(*start = q_ct1.ext_oneapi_submit_barrier()));
  if (0)
    MY_ERROR_CHECKER(hipEventRecord(start, 0));

// CHECK:  MY_ERROR_CHECKER(DPCT_CHECK_ERROR(*start = q_ct1.ext_oneapi_submit_barrier()));
  MY_ERROR_CHECKER(hipEventRecord(start));

  // kernel call without sync
// CHECK:  DPCT1049:{{[0-9a-f]+}}: The work-group size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the work-group size if needed.
// CHECK-NEXT:  */
// CHECK-NEXT:  q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          kernelFunc();
// CHECK-NEXT:        });
  kernelFunc<<<blocks,threads>>>();

// CHECK:  *stop = q_ct1.ext_oneapi_submit_barrier();
// CHECK-NEXT:  stop->wait_and_throw();
// CHECK-NEXT:  elapsed_time = (stop->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time, start, stop);

// CHECK: MY_ERROR_CHECKER(DPCT_CHECK_ERROR(elapsed_time = (stop->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f));
  MY_ERROR_CHECKER(hipEventElapsedTime(&elapsed_time, start, stop));

}

void foo() {
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  float elapsed_time;

  hipDeviceSynchronize();

  int blocks = 32, threads = 32;

// CHECK:  MY_CHECKER(DPCT_CHECK_ERROR(*start = q_ct1.ext_oneapi_submit_barrier()));
  MY_CHECKER(hipEventRecord(start, 0));
  kernelFunc<<<blocks,threads>>>();
// CHECK: MY_CHECKER(DPCT_CHECK_ERROR(*stop = q_ct1.ext_oneapi_submit_barrier()));
  MY_CHECKER(hipEventRecord(stop, 0));

  hipEventSynchronize(stop);

// CHECK: MY_CHECKER(DPCT_CHECK_ERROR(elapsed_time = (stop->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f));
  MY_CHECKER(hipEventElapsedTime(&elapsed_time, start, stop));

  hipEventDestroy(start);
  hipEventDestroy(stop);

  {
  // CHECK: dpct::err0 err = DPCT_CHECK_ERROR(*start = q_ct1.ext_oneapi_submit_barrier());
    hipError_t err = hipEventRecord(start, 0);
  // CHECK: err = DPCT_CHECK_ERROR(*stop = q_ct1.ext_oneapi_submit_barrier());
    err = hipEventRecord(stop, 0);
    if (hipSuccess != err) {
      printf("%s\n", hipGetErrorString( err));
    }
  }
}

void fun(int) {}

void bar() {
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  float elapsed_time;

  hipDeviceSynchronize();

  int blocks = 32, threads = 32;

// CHECK: fun(DPCT_CHECK_ERROR(*start = q_ct1.ext_oneapi_submit_barrier()));
  fun(hipEventRecord(start, 0));
  kernelFunc<<<blocks,threads>>>();
// CHECK: fun(DPCT_CHECK_ERROR(*stop = q_ct1.ext_oneapi_submit_barrier()));
  fun(hipEventRecord(stop, 0));

  hipEventSynchronize(stop);
// CHECK: fun(DPCT_CHECK_ERROR(elapsed_time = (stop->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f));
  fun(hipEventElapsedTime(&elapsed_time, start, stop));

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

struct Node {
 // CHECK: dpct::event_ptr start;
 hipEvent_t start;
 // CHECK: dpct::event_ptr end;
 hipEvent_t end;
 // CHECK: dpct::event_ptr *ev[100];
 hipEvent_t *ev[100];
 // CHECK: dpct::event_ptr events[100];
 hipEvent_t events[100];
 // CHECK: dpct::event_ptr *p_events;
 hipEvent_t *p_events;
};

void foo2(Node *n) {
  float elapsed_time;

// CHECK: *n->start = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(n->start, 0);
// CHECK: *n->start = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(n->start, 0);
  // do something
// CHECK: *n->end = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(n->end, 0);
// CHECK: *n->end = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(n->end, 0);
// CHECK: elapsed_time = (n->end->get_profiling_info<sycl::info::event_profiling::command_end>() - n->start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
  hipEventElapsedTime(&elapsed_time, n->start, n->end);
  {
    int errorCode;
  // CHECK: MY_CHECKER(DPCT_CHECK_ERROR(*n->start = q_ct1.ext_oneapi_submit_barrier()));
    MY_CHECKER(hipEventRecord(n->start, 0));
  // CHECK: errorCode = DPCT_CHECK_ERROR(*n->start = q_ct1.ext_oneapi_submit_barrier());
    errorCode = hipEventRecord(n->start, 0);
  }

  Node node;
// CHECK: *node.start = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(node.start, 0);
// CHECK: *node.start = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(node.start, 0);
  // do something
// CHECK: *node.end = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(node.end, 0);
// CHECK: *node.end = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(node.end, 0);
// CHECK: elapsed_time = (node.end->get_profiling_info<sycl::info::event_profiling::command_end>() - node.start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
  hipEventElapsedTime(&elapsed_time, node.start, node.end);
  {
    int errorCode;
  // CHECK: MY_CHECKER(DPCT_CHECK_ERROR(*node.start = q_ct1.ext_oneapi_submit_barrier()));
    MY_CHECKER(hipEventRecord(node.start, 0));
  // CHECK: errorCode = DPCT_CHECK_ERROR(*node.start = q_ct1.ext_oneapi_submit_barrier());
    errorCode = hipEventRecord(node.start, 0);
  }

  {
  // CHECK: *node.events[0] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(node.events[0]);
  // CHECK: *node.events[0] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(node.events[0]);
  // CHECK: *node.events[23] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(node.events[23]);
  // CHECK: *node.events[23] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(node.events[23]);
  // CHECK: elapsed_time = (node.events[23]->get_profiling_info<sycl::info::event_profiling::command_end>() - node.events[0]->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
    hipEventElapsedTime(&elapsed_time, node.events[0], node.events[23]);
  }

  {
  // CHECK: **node.ev[0] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*node.ev[0]);
  // CHECK: **node.ev[0] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*node.ev[0]);
  // CHECK: **node.ev[23] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*node.ev[23]);
  // CHECK: **node.ev[23] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*node.ev[23]);
  // CHECK: elapsed_time = (*node.ev[23]->get_profiling_info<sycl::info::event_profiling::command_end>() - *node.ev[0]->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
    hipEventElapsedTime(&elapsed_time, *node.ev[0], *node.ev[23]);
  }

  {
  // CHECK: **(&node)->ev[0] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*(&node)->ev[0]);
  // CHECK: **(&node)->ev[0] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*(&node)->ev[0]);
  // CHECK: **(&node)->ev[23] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*(&node)->ev[23]);
  // CHECK: **(&node)->ev[23] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*(&node)->ev[23]);
  // CHECK:  elapsed_time = (*(&node)->ev[23]->get_profiling_info<sycl::info::event_profiling::command_end>() - *(&node)->ev[0]->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
    hipEventElapsedTime(&elapsed_time, *(&node)->ev[0], *(&node)->ev[23]);
  }

  {
  // CHECK: *n->p_events[0] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(n->p_events[0]);
  // CHECK: *n->p_events[1] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(n->p_events[1]);
  // CHECK:  *n->p_events[2] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(n->p_events[2]);
  // CHECK:  *n->p_events[3] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(n->p_events[3]);
  }
}
