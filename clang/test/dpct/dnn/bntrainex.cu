// RUN: dpct -in-root %S -out-root %T/bntrainex %S/bntrainex.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/bntrainex/bntrainex.dp.cpp --match-full-lines %s
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);

    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, scalebiasTensor;
    cudnnCreate(&handle);

    hipStream_t stream1;
    hipStreamCreate(&stream1);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateTensorDescriptor(&scalebiasTensor);

    int in = 2, ic = 4, ih = 5, iw = 5;
    int on = 2, oc = 4, oh = 5, ow = 5;
    int sbn = 1, sbc = 4, sbh = 1, sbw = 1;
    int ele_num = in* ic * ih * iw;
    int oele_num = on* oc * oh * ow;
    int sele_num = sbn*sbc * sbh * sbw;
    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, in, ic, ih, iw);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, on, oc, oh, ow);
    cudnnSetTensor4dDescriptor(scalebiasTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, sbn, sbc, sbh, sbw);

    int save = 1;
    float *data, *out, *scale, *bias, *rmean, *rvar, *smean, *svar, *z;
    std::vector<float> host_data(ele_num, 1.0f);
    std::vector<float> host_z(oele_num, 1.0f);
    std::vector<float> host_out(oele_num, 0.0f);
    std::vector<float> host_scale(sele_num, 1.0f);
    std::vector<float> host_bias(sele_num, 0.0f);
    std::vector<float> host_rmean(sele_num, 0.0f);
    std::vector<float> host_rvar(sele_num, 0.0f);
    std::vector<float> host_smean(save * sele_num, 0.0f);
    std::vector<float> host_svar(save * sele_num, 0.0f);

    for(int i = 0; i < ele_num; i++) {
        host_data[i] =  i + 4.f;
        host_out[i] = 1.f;
        host_z[i] = 10;
    }
    for(int i = 0; i < sele_num; i++) {
        host_scale[i] = i;
        host_bias[i] = i;
        host_rmean[i] = i;
        host_rvar[i] = i;
        host_smean[i] = i;
        host_svar[i] = i;
    }

    hipMalloc(&data, sizeof(float) * ele_num);
    hipMalloc(&z, sizeof(float) * oele_num);
    hipMalloc(&out, sizeof(float) * oele_num);
    hipMalloc(&scale, sizeof(float) * sele_num);
    hipMalloc(&bias, sizeof(float) * sele_num);
    hipMalloc(&rmean, sizeof(float) * sele_num);
    hipMalloc(&rvar, sizeof(float) * sele_num);
    hipMalloc(&smean, sizeof(float) * save*sele_num);
    hipMalloc(&svar, sizeof(float)  * save*sele_num);

    hipMemcpy(data, host_data.data(), sizeof(float) * ele_num, hipMemcpyHostToDevice);
    hipMemcpy(z, host_z.data(), sizeof(float) * oele_num, hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), sizeof(float) * oele_num, hipMemcpyHostToDevice);
    hipMemcpy(scale, host_scale.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(bias, host_bias.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(rmean, host_rmean.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(rvar, host_rvar.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(smean, host_smean.data(),  sizeof(float) * save * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(svar, host_svar.data(), sizeof(float) * save * sele_num, hipMemcpyHostToDevice);

    float alpha = 1.f, beta = 0.f, eps = 1.f;
    double factor = 0.5f;
    // CHECK: dpct::dnnl::activation_desc ActivationDesc;
    // CHECK: ActivationDesc.set(dnnl::algorithm::eltwise_relu_use_dst_for_bwd, 0.0f);
    cudnnActivationDescriptor_t ActivationDesc;
    cudnnCreateActivationDescriptor(&ActivationDesc);

    cudnnSetActivationDescriptor(ActivationDesc, CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0.0f);
    
    float *workspace, *reservespace;
    size_t workspace_size, reservespace_size;
    // CHECK: workspace_size = 0;
    cudnnGetBatchNormalizationForwardTrainingExWorkspaceSize(
        handle, 
        //CUDNN_BATCHNORM_PER_ACTIVATION, 
        CUDNN_BATCHNORM_SPATIAL, 
        //CUDNN_BATCHNORM_OPS_BN_ACTIVATION, 
        CUDNN_BATCHNORM_OPS_BN,
        //CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION, 
        dataTensor,
        outTensor, 
        outTensor, 
        scalebiasTensor, 
        ActivationDesc, 
        &workspace_size);
    // CHECK: reservespace_size = handle.get_batch_normalization_workspace_size(dpct::dnnl::batch_normalization_ops::none, dataTensor);
    cudnnGetBatchNormalizationTrainingExReserveSpaceSize(
        handle, 
        //CUDNN_BATCHNORM_PER_ACTIVATION, 
        CUDNN_BATCHNORM_SPATIAL, 
        //UDNN_BATCHNORM_OPS_BN_ACTIVATION, 
        CUDNN_BATCHNORM_OPS_BN,
        //CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION, 
        ActivationDesc, 
        dataTensor, 
        &reservespace_size);
    
    hipMalloc(&workspace, workspace_size);
    hipMalloc(&reservespace, reservespace_size);
    // CHECK: auto status = DPCT_CHECK_ERROR(handle.async_batch_normalization_forward_training(dpct::dnnl::batch_normalization_mode::spatial, dpct::dnnl::batch_normalization_ops::none, ActivationDesc, eps, factor, alpha, dataTensor, data, beta, outTensor, out, outTensor, z, scalebiasTensor, scale, bias, rmean, rvar, smean, svar, reservespace_size, reservespace));
    auto status = cudnnBatchNormalizationForwardTrainingEx(
        handle, 
        //CUDNN_BATCHNORM_PER_ACTIVATION, 
        CUDNN_BATCHNORM_SPATIAL, 
        //CUDNN_BATCHNORM_OPS_BN_ACTIVATION, 
        CUDNN_BATCHNORM_OPS_BN,
        //CUDNN_BATCHNORM_OPS_BN_ADD_ACTIVATION,
        &alpha,
        &beta,
        dataTensor,
        data,
        outTensor,
        z,
        outTensor,
        out,
        scalebiasTensor,
        scale,
        bias,
        factor,
        rmean,
        rvar,
        eps,
        smean,
        svar,
        ActivationDesc,
        workspace,
        workspace_size,
        reservespace,
        reservespace_size
    );

    hipDeviceSynchronize();
    hipMemcpy(host_out.data(), out, sizeof(float) * oele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_smean.data(), smean,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_svar.data(), svar,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_rmean.data(), rmean,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_rvar.data(), rvar,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);

    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
    return 0;
}