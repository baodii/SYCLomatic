// RUN: dpct -in-root %S -out-root %T/reorder %S/reorder.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/reorder/reorder.dp.cpp --match-full-lines %s

// CHECK: #include <dpct/dnnl_utils.hpp>
// CHECK: #include <sycl/sycl.hpp>
// CHECK: #include <dpct/dpct.hpp>
// CHECK: #include <iostream>
// CHECK: #include <vector>
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

// CHECK: template <dpct::library_data_t T>
// CHECK: struct dt_trait {
// CHECK:     typedef void type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_float> {
// CHECK:     typedef float type;
// CHECK: };
// CHECK: template <>
// CHECK: /*
// CHECK: DPCT1007:{{[0-9]+}}: Migration of CUDNN_DATA_DOUBLE is not supported.
// CHECK: */
// CHECK: struct dt_trait<CUDNN_DATA_DOUBLE> {
// CHECK:     typedef double type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_int32> {
// CHECK:     typedef int type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_half> {
// CHECK:     typedef float type;
// CHECK: };

template<cudnnDataType_t T>
struct dt_trait{
    typedef void type;
};
template<>
struct dt_trait<CUDNN_DATA_FLOAT>{
    typedef float type;
};
template<>
struct dt_trait<CUDNN_DATA_DOUBLE>{
    typedef double type;
};
template<>
struct dt_trait<CUDNN_DATA_INT32>{
    typedef int type;
};
template<>
struct dt_trait<CUDNN_DATA_HALF>{
    typedef float type;
};


template<cudnnDataType_t T, typename HT = typename dt_trait<T>::type>
void test() {

    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor;

    cudnnCreate(&handle);

    hipStream_t stream1;
    hipStreamCreate(&stream1);
    cudnnSetStream(handle, stream1);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    int n = 1, c = 2, h = 5, w = 5;
    int ele_num = n * c * h * w;

    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NHWC, T, n, c, h, w);

    HT *data, *out;
    std::vector<HT> host_data(ele_num, 0);
    std::vector<HT> host_out(ele_num, 0);

    for(int i = 0; i < ele_num; i++) {
        host_data[i] = i;
        host_out[i] = 0;
    }


    hipMalloc(&data, ele_num * sizeof(HT));
    hipMalloc(&out, ele_num * sizeof(HT));

    hipMemcpy(data, host_data.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);


    float alpha = 3.f, beta = 1.f;
    //CHECK: auto s = DPCT_CHECK_ERROR(handle.async_reorder(alpha, dataTensor, data, beta, outTensor, out));
    auto s = cudnnTransformTensor(handle, &alpha, dataTensor, data, &beta, outTensor, out);

    hipMemcpy(host_out.data(), out, ele_num * sizeof(HT), hipMemcpyDeviceToHost);

    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
}

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    hipSetDevice(1);
    
    test<CUDNN_DATA_FLOAT>();
       
    return 0;
}