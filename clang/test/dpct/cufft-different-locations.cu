// RUN: dpct --format-range=none --usm-level=none -out-root %T/cufft-different-locations %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cufft-different-locations/cufft-different-locations.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

size_t* work_size;
int odist;
int ostride;
int * onembed;
int idist;
int istride;
int* inembed;
int * n;
double* odata;
double2* idata;

#define HANDLE_CUFFT_ERROR( err ) (CufftHandleError( err, __FILE__, __LINE__ ))
static void CufftHandleError( hipfftResult err, const char *file, int line ) {
  if (err != HIPFFT_SUCCESS) {
    fprintf(stderr, "Cufft error in file '%s' in line %i : %s.\n",
            __FILE__, __LINE__, "error" );
  }
}

int main() {
  hipfftHandle plan1;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:int res1 = DPCT_CHECK_ERROR(plan1->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size));
  hipfftResult res1 = hipfftMakePlanMany(plan1, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
  //CHECK:int res2 = DPCT_CHECK_ERROR((plan1->compute<sycl::double2, double>(idata, odata, dpct::fft::fft_direction::backward)));
  hipfftResult res2 = hipfftExecZ2D(plan1, idata, odata);

  hipfftHandle plan2;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:res1 = DPCT_CHECK_ERROR(plan2->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size));
  res1 = hipfftMakePlanMany(plan2, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
  //CHECK:res2 = DPCT_CHECK_ERROR((plan2->compute<sycl::double2, double>(idata, odata, dpct::fft::fft_direction::backward)));
  res2 = hipfftExecZ2D(plan2, idata, odata);

  hipfftHandle plan3;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:HANDLE_CUFFT_ERROR(DPCT_CHECK_ERROR(plan3->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size)));
  HANDLE_CUFFT_ERROR(hipfftMakePlanMany(plan3, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size));
  //CHECK:HANDLE_CUFFT_ERROR(DPCT_CHECK_ERROR((plan3->compute<sycl::double2, double>(idata, odata, dpct::fft::fft_direction::backward))));
  HANDLE_CUFFT_ERROR(hipfftExecZ2D(plan3, idata, odata));

  hipfftHandle plan4;
  hipfftHandle plan5;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:if(DPCT_CHECK_ERROR(plan4->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size))) {
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:} else if (DPCT_CHECK_ERROR(plan5->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size))) {
  //CHECK-NEXT:}
  if(hipfftMakePlanMany(plan4, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)) {
  } else if (hipfftMakePlanMany(plan5, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)) {
  }
  //CHECK:if (DPCT_CHECK_ERROR((plan4->compute<sycl::double2, double>(idata, odata, dpct::fft::fft_direction::backward)))) {
  //CHECK-NEXT:} else if(DPCT_CHECK_ERROR((plan5->compute<sycl::double2, double>(idata, odata, dpct::fft::fft_direction::backward)))) {
  //CHECK-NEXT:}
  if (hipfftExecZ2D(plan4, idata, odata)) {
  } else if(hipfftExecZ2D(plan5, idata, odata)) {
  }

  hipfftHandle plan6;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:if(int res = DPCT_CHECK_ERROR(plan6->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size))) {
  //CHECK-NEXT:}
  if(hipfftResult res = hipfftMakePlanMany(plan6, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)) {
  }
  //CHECK:if(int res = DPCT_CHECK_ERROR((plan6->compute<sycl::double2, double>(idata, odata, dpct::fft::fft_direction::backward)))) {
  //CHECK-NEXT:}
  if(hipfftResult res = hipfftExecZ2D(plan6, idata, odata)) {
  }

  hipfftHandle plan7;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:for (plan7->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size);;) {
  //CHECK-NEXT:}
  for (hipfftMakePlanMany(plan7, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);;) {
  }
  //CHECK:for (plan7->compute<sycl::double2, double>(idata, odata, dpct::fft::fft_direction::backward);;) {
  //CHECK-NEXT:}
  for (hipfftExecZ2D(plan7, idata, odata);;) {
  }

  hipfftHandle plan8;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:for (;DPCT_CHECK_ERROR(plan8->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size));) {
  //CHECK-NEXT:}
  for (;hipfftMakePlanMany(plan8, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);) {
  }

  //CHECK:for (;DPCT_CHECK_ERROR((plan8->compute<sycl::double2, double>(idata, odata, dpct::fft::fft_direction::backward)));) {
  //CHECK-NEXT:}
  for (;hipfftExecZ2D(plan8, idata, odata);) {
  }

  hipfftHandle plan9;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:while (DPCT_CHECK_ERROR(plan9->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size)) != 0) {
  //CHECK-NEXT:}
  while (hipfftMakePlanMany(plan9, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size) != 0) {
  }

  //CHECK:while (DPCT_CHECK_ERROR((plan9->compute<sycl::double2, double>(idata, odata, dpct::fft::fft_direction::backward))) != 0) {
  //CHECK-NEXT:}
  while (hipfftExecZ2D(plan9, idata, odata) != 0) {
  }

  hipfftHandle plan10;
  //CHECK:do {
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:} while (DPCT_CHECK_ERROR(plan10->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size)));
  do {
  } while (hipfftMakePlanMany(plan10, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size));
  //CHECK:do {
  //CHECK-NEXT:} while (DPCT_CHECK_ERROR((plan10->compute<sycl::double2, double>(idata, odata, dpct::fft::fft_direction::backward))));
  do {
  } while (hipfftExecZ2D(plan10, idata, odata));

  hipfftHandle plan11;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:switch (int stat = DPCT_CHECK_ERROR(plan11->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size))){
  //CHECK-NEXT:}
  switch (int stat = hipfftMakePlanMany(plan11, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)){
  }

  //CHECK:switch (int stat = DPCT_CHECK_ERROR((plan11->compute<sycl::double2, double>(idata, odata, dpct::fft::fft_direction::backward)))){
  //CHECK-NEXT:}
  switch (int stat = hipfftExecZ2D(plan11, idata, odata)){
  }
  return 0;
}

hipfftResult foo1(hipfftHandle plan) {
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:return DPCT_CHECK_ERROR(plan->commit(&dpct::get_out_of_order_queue(), 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size));
  return hipfftMakePlanMany(plan, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
}

hipfftResult foo2(hipfftHandle plan) {
  //CHECK:return DPCT_CHECK_ERROR((plan->compute<sycl::double2, double>(idata, odata, dpct::fft::fft_direction::backward)));
  return hipfftExecZ2D(plan, idata, odata);
}

hipfftResult foo3(hipfftHandle plan) {
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(&dpct::get_out_of_order_queue(), 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size);
  hipfftMakePlanMany(plan, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
}

hipfftResult foo4(hipfftHandle plan) {
  //CHECK:plan->compute<sycl::double2, double>(idata, odata, dpct::fft::fft_direction::backward);
  hipfftExecZ2D(plan, idata, odata);
}

