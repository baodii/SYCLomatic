// RUN: dpct --format-range=none -out-root %T/cufft-refine-setstream %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cufft-refine-setstream/cufft-refine-setstream.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void foo1() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:plan->set_queue(s);
  //CHECK-NEXT:plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  hipfftSetStream(plan, s);
  hipfftExecR2C(plan, (float*)iodata, iodata);
}


#define cufftCheck(stmt) \
do {                                           \
  hipfftResult err = stmt;                                               \
  if (err != HIPFFT_SUCCESS) {                                           \
  }                                                                     \
} while(0)
void foo2() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:cufftCheck(DPCT_CHECK_ERROR(plan->set_queue(s)));
  //CHECK-NEXT:cufftCheck(DPCT_CHECK_ERROR((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward))));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  cufftCheck(hipfftSetStream(plan, s));
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
}
#undef cufftCheck



#define HANDLE_CUFFT_ERROR( err ) (CufftHandleError( err, __FILE__, __LINE__ ))
static void CufftHandleError( hipfftResult err, const char *file, int line )
{
    if (err != HIPFFT_SUCCESS)
    {
      int a = err;
    }
}

void foo3() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:HANDLE_CUFFT_ERROR(DPCT_CHECK_ERROR(plan->set_queue(s)));
  //CHECK-NEXT:HANDLE_CUFFT_ERROR(DPCT_CHECK_ERROR((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward))));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  HANDLE_CUFFT_ERROR(hipfftSetStream(plan, s));
  HANDLE_CUFFT_ERROR(hipfftExecR2C(plan, (float*)iodata, iodata));
}
#undef HANDLE_CUFFT_ERROR


void foo4() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:int err = DPCT_CHECK_ERROR(plan->set_queue(s));
  //CHECK-NEXT:err = DPCT_CHECK_ERROR((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward)));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  hipfftResult err = hipfftSetStream(plan, s);
  err = hipfftExecR2C(plan, (float*)iodata, iodata);
}


static inline void CUFFT_CHECK(hipfftResult error)
{
  if (error != HIPFFT_SUCCESS) {
  }
}

void foo5() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:CUFFT_CHECK(DPCT_CHECK_ERROR(plan->set_queue(s)));
  //CHECK-NEXT:CUFFT_CHECK(DPCT_CHECK_ERROR((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward))));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  CUFFT_CHECK(hipfftSetStream(plan, s));
  CUFFT_CHECK(hipfftExecR2C(plan, (float*)iodata, iodata));
}


#define MY_ERROR_CHECKER(CALL) my_error_checker((CALL), #CALL)
template <typename T>
void my_error_checker(T ReturnValue, char const *const FuncName) {}

void foo6() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:MY_ERROR_CHECKER(DPCT_CHECK_ERROR(plan->set_queue(s)));
  //CHECK-NEXT:MY_ERROR_CHECKER(DPCT_CHECK_ERROR((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward))));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  MY_ERROR_CHECKER(hipfftSetStream(plan, s));
  MY_ERROR_CHECKER(hipfftExecR2C(plan, (float*)iodata, iodata));
}
#undef MY_ERROR_CHECKER


#define CHECK_CUFFT(call)                                                      \
{                                                                              \
  hipfftResult err;                                                           \
  if ( (err = (call)) != HIPFFT_SUCCESS)                                      \
  {                                                                          \
  }                                                                          \
}
void foo7() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:CHECK_CUFFT(DPCT_CHECK_ERROR(plan->set_queue(s)));
  //CHECK-NEXT:CHECK_CUFFT(DPCT_CHECK_ERROR((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward))));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  CHECK_CUFFT(hipfftSetStream(plan, s));
  CHECK_CUFFT(hipfftExecR2C(plan, (float*)iodata, iodata));
}
#undef CHECK_CUFFT

#define cufftCheck(stmt) \
do {                                           \
  hipfftResult err;                                                      \
  if ( (err = (stmt)) != HIPFFT_SUCCESS) {                               \
  }                                                                     \
} while(0)
void foo8() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:cufftCheck(DPCT_CHECK_ERROR(plan->set_queue(s)));
  //CHECK-NEXT:cufftCheck(DPCT_CHECK_ERROR((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward))));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  cufftCheck(hipfftSetStream(plan, s));
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
}


void foo9() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s1, s2;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:cufftCheck(DPCT_CHECK_ERROR((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward))));
  //CHECK-NEXT:cufftCheck(DPCT_CHECK_ERROR(plan->set_queue(s1)));
  //CHECK-NEXT:cufftCheck(DPCT_CHECK_ERROR((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward))));
  //CHECK-NEXT:cufftCheck(DPCT_CHECK_ERROR(plan->set_queue(s2)));
  //CHECK-NEXT:cufftCheck(DPCT_CHECK_ERROR((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward))));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
  cufftCheck(hipfftSetStream(plan, s1));
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
  cufftCheck(hipfftSetStream(plan, s2));
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
}

// This case needs manual fix
void foo10(bool flag) {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:cufftCheck(DPCT_CHECK_ERROR((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward))));
  //CHECK-NEXT:if (flag) {
  //CHECK-NEXT:  cufftCheck(DPCT_CHECK_ERROR(plan->set_queue(s)));
  //CHECK-NEXT:}
  //CHECK-NEXT:cufftCheck(DPCT_CHECK_ERROR((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward))));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
  if (flag) {
    cufftCheck(hipfftSetStream(plan, s));
  }
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
}
#undef cufftCheck

void foo11(bool flag) {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:if (flag)
  //CHECK-NEXT:  plan->set_queue(s);
  //CHECK-NEXT:plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  if (flag)
    hipfftSetStream(plan, s);
  hipfftExecR2C(plan, (float*)iodata, iodata);
}

void foo12(hipfftHandle plan2) {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:plan->set_queue(s);
  //CHECK-NEXT:plan = plan2;
  //CHECK-NEXT:plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  hipfftSetStream(plan, s);
  plan = plan2;
  hipfftExecR2C(plan, (float*)iodata, iodata);
}

void changeHandle(hipfftHandle &p);

void foo13() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:plan->set_queue(s);
  //CHECK-NEXT:changeHandle(plan);
  //CHECK-NEXT:plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  hipfftSetStream(plan, s);
  changeHandle(plan);
  hipfftExecR2C(plan, (float*)iodata, iodata);
}

void foo14() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:plan->set_queue(s);
  //CHECK-NEXT:plan = dpct::fft::fft_engine::create();
  //CHECK-NEXT:plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward);
  //CHECK-NEXT:dpct::fft::fft_engine::destroy(plan);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  hipfftSetStream(plan, s);
  hipfftCreate(&plan);
  hipfftExecR2C(plan, (float*)iodata, iodata);
  hipfftDestroy(plan);
}
