// Option: --use-dpcpp-extensions=intel_device_math
// Option: --use-experimental-features=bfloat16_math_functions

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half2 h, __hip_bfloat162 b) {
  // Start
  h2log(h /*__half2*/);
  h2log(b /*__nv_bfloat162*/);
  // End
}
