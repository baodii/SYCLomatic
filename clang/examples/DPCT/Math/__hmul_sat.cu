// Option: --use-dpcpp-extensions=intel_device_math

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half h1, __half h2, __hip_bfloat16 b1, __hip_bfloat16 b2) {
  // Start
  __hmul_sat(h1 /*__half*/, h2 /*__half*/);
  __hmul_sat(b1 /*__nv_bfloat16*/, b2 /*__nv_bfloat16*/);
  // End
}
