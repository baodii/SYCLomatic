// Option: --use-dpcpp-extensions=intel_device_math

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half h1, __half h2, __hip_bfloat16 b1, __hip_bfloat16 b2,
                     int i1, int i2) {
  // Start
  __hadd(h1 /*__half*/, h2 /*__half*/);
  __hadd(b1 /*__nv_bfloat16*/, b2 /*__nv_bfloat16*/);
  __hadd(i1 /*int*/, i2 /*int*/);
  // End
}
